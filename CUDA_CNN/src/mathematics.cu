#include "hip/hip_runtime.h"
/*
 * mathematics.cpp
 *
 *  Created on: 05.12.2017
 *      Author: Florian
 */

#include "mathematics.h"
#include "math.h"

namespace mathematics {


__device__ float sigmoid_once(float in)
{
	double temp = exp(in);
	return (float)(temp / (1+temp));
}

__device__ float sigmoid_backward_derivated_once(float activation)
{
	return activation * (1 - activation);
}

__device__ void sigmoid(float *in, float *out, int size)
{
	for(; size>0; size--, in++, out++)
	{
		*out = sigmoid_once(*in);
	}
}

__device__ void sigmoid_backward_derivated(float *activation, float *derivatives, int size)
{
	for(; size>0; size--, activation++, derivatives++)
	{
		*derivatives = sigmoid_backward_derivated_once(*activation);
	}
}


__device__ void softmax(float *in, float *out, int size)
{
	double sum=0;
	for(int i=0; i<size; i++)
	{
		sum += exp(in[i]);
	}
	for(int i=0; i<size; i++)
	{
		out[i] = (float)(exp(in[i]) / sum);
	}
}


__device__ float cross_entropy(float *calculated, float *expected, int size)
{
	double sum=0;
	for(; size>0; size--, expected++, calculated++)
	{
		sum += - (*expected) * log(*calculated);
	}
	return (float)sum;
}

__device__ float get_cost(float *output, float *labels, int size)
{
	float *normalized;
	float ret;
	normalized = new float[size];
	softmax(output, normalized, size);
	ret = cross_entropy(normalized, labels, size);
	delete[] normalized;
	return ret;
}

/* quadratic cost function derivated */
__device__ void get_cost_derivatives(float *output, float *labels, float *derivatives, int size)
{
	for(; size>0; size--, output++, labels++, derivatives++)
	{
		*derivatives = *output - *labels;
	}
}

__device__ void hadamard_product(float* a, float* b, float* out, int size)
{
	int index = threadIdx.x;
	int stride = blockDim.x;

	for(int i = index; i < size; i+=stride)
	{
		out[i] = a[i] * b[i];
	}
	__syncthreads();
}

} /* namespace mathematics */
